
#include <hip/hip_runtime.h>
#include <iostream>
//#include <math.h>
//#include <queue>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
//#include <fstream>

#define SIZE 32

typedef struct{
  int n;
  bool a[SIZE][SIZE];
} _ADJ;

__host__ void print(_ADJ t){
  std::cout<<"n="<<t.n<<": "<<std::endl;
  for(int i=0;i<t.n;i++){
    std::cout<<"(";
    for(int j=0;j<t.n;j++){
      std::cout<<t.a[i][j];
    }
    std::cout<<")"<<std::endl;
  }
  std::cout<<std::endl;
}

__host__ __device__  void copy(_ADJ *to, _ADJ *from){
  to->n=from->n;
  for(int i=0;i<from->n;i++){
     for(int j=0;j<from->n;j++){
       to->a[i][j]=from->a[i][j];
     }
  }
}


__host__ __device__ int BadTriples(_ADJ t){
  int n=t.n;	    
  int count=0;
  for(int i=0; i<n-2; i++){
    for(int j=i+1; j<n-1;j++){
      for(int k=j+1; k <n; k++){
	bool b[8];
	int s;
	for(s=0;s<8;s++){
	  b[s]=0;
	}
	for(int l=0;l<n;l++){
	  if(l!=i && l!=j && l!=k){
	    s=0;
	    if(t.a[i][l]) s=s+4;
	    if(t.a[j][l]) s=s+2;
	    if(t.a[k][l]) s=s+1;
	    b[s]=1;
	  }
	}
	if(not (b[0] && b[1] && b[2] && b[3] && b[4] && b[5] && b[6] && b[7])) count++;                 
      }
    }
  }
  return count;
}
     

__global__ void VecBadTriples(_ADJ* t, int* y, int N){
  int i = blockDim.x * blockIdx.x + threadIdx.x; 
  if (i < N)  {
    y[i]=BadTriples(t[i]);
  }
}


void checkCUDAError(const char *msg){

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
              hipGetErrorString( err) );
      exit(EXIT_FAILURE);
    }                         
}


int main(int argc, char* argv[]){

  if(argc <2){
    std::cout<<"Usage:"<<std::endl;
    std::cout<<argv[0]<<" n target"<<std::endl;
    return 0;
  }

  int n;
  n=atoi(argv[1]);
  
  int target;
  target=atoi(argv[2]);
 
  
  _ADJ cur;
  _ADJ tmp;
  
  
  srand(time(0));
  cur.n=n;
  for(int i=0;i<n;i++){
    for(int j=i;j<n;j++){
      
      
      if(rand()%2==0){
	cur.a[i][j]=cur.a[j][i]=0;
      }
      else{
	cur.a[i][j]=cur.a[j][i]=1;
      }
    }
  }
 
  int bt_cur= BadTriples(cur);
  int bt_tmp;
  int bt_next=bt_cur;
  std::cout<<"Bad trples ="<<bt_cur<<std::endl;
  
  int e=n*(n-1)/2;
  int N=e*(e-1)/2;
  
  std::cout<<"e="<<e<<" N="<<N<<std::endl;

  int memsize = N * sizeof(_ADJ);
  _ADJ * t0 = (_ADJ*) malloc(memsize);

  _ADJ * dev_t0;
  hipMalloc((void**)&dev_t0, memsize);
  

  int * bt = (int *) malloc(N * sizeof(int));
  int * dev_bt;
  hipMalloc((void**)&dev_bt, N* sizeof(int));
  checkCUDAError("hipMalloc dev_bt");

  while(bt_cur>target){
    std::cout<<"Reshaffling..."<<std::endl;
    //flip a small portion 1/n of edges
    for(int i=0;i<n;i++){ 
    for(int j=i;j<n;j++){ 
      cur.n=n;
      if(rand()%(n)==0){
	cur.a[j][i]=1-cur.a[j][i]; //flip
	cur.a[i][j]=cur.a[j][i];
      }
    }
    }
    
    bt_cur= BadTriples(cur);
    copy(&tmp, &cur);
    bt_tmp= bt_cur;
    
    while(true){
      
    std::cout<<"Enter while loop..."<<std::endl;
    std::cout<<"bad triples="<<bt_cur<<std::endl;
    int s=0;
    for(int i=0;i<n-1;i++){
      for(int j=i+1; j<n;j++){
	for(int k=i;k<n-1; k++){
	  for(int l=k+1;l<n;l++){
	    if(k>i || l>j){
	      for(int p=k;p<n-1;p++){
		for(int q=p+1;q<n;q++){
		  if(p>k || q>l){		  
		    copy(&t0[s], &cur); 
		    t0[s].a[i][j]=1-t0[s].a[i][j]; //flip edge ij
		    t0[s].a[j][i]=t0[s].a[i][j];
		    
		    t0[s].a[k][l]=1-t0[s].a[k][l]; //flip edge kl
		    t0[s].a[l][k]=t0[s].a[k][l];
		    
		    t0[s].a[p][q]=1-t0[s].a[p][q]; //flip edge pq
		    t0[s].a[q][p]=t0[s].a[p][q];
		    s++;

		    if(s==N){
		      hipMemcpy(dev_t0, t0, memsize,  hipMemcpyHostToDevice);
		      checkCUDAError("hipMemcpyHostToDevice");
  
		      VecBadTriples<<<(N+1023)/1024, 1024>>>(dev_t0, dev_bt, N);
		      checkCUDAError("kernel VecBadTriples");

		      hipMemcpy(bt, dev_bt, sizeof(int)*N,  hipMemcpyDeviceToHost);
		      checkCUDAError("hipMemcpyDeviceToHost");

		      bt_next=bt_tmp;
		      int min_x=-1;
		      for(int x=0; x<N;x++){
			if(bt[x]<bt_next){
			  min_x=x;
			  bt_next=bt[x];
			}
		      }

		      if(min_x>0){
			copy(&tmp, &t0[min_x]);
			bt_tmp=bt_next;
			//std::cout<<"bad triples="<<bt_tmp<<std::endl;
		      }
		      s=0;
		    }
		  }
		}
	      }
	    }
	  }
	}
      }
    }

    if(s>0){
      hipMemcpy(dev_t0, t0, memsize,  hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpyHostToDevice");
    
      VecBadTriples<<<(s+1023)/1024, 1024>>>(dev_t0, dev_bt, s);
      checkCUDAError("kernel VecBadTriples");

      hipMemcpy(bt, dev_bt, sizeof(int)*s,  hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpyDeviceToHost");

      bt_next=bt_tmp;
      int min_x=-1;
      for(int x=0;x<s;x++){
    
	if(bt[x]<bt_next){
	  min_x=x;
	  bt_next=bt[x];
	}
      }

      if(min_x>0){
	copy(&tmp, &t0[min_x]);
	bt_tmp=bt_next;
	//std::cout<<"bad triples="<<bt_tmp<<std::endl;
      }
    }
    
    if(bt_tmp<bt_cur){
      copy(&cur, &tmp);
      bt_cur=bt_tmp;
    }
    else{
      break;
    }
    }
    std::cout<<"bad triples="<<bt_cur<<std::endl;
  }

  print(cur);
  std::cout<<"Bad trples ="<<BadTriples(cur)<<std::endl;

  hipFree(dev_t0);
  hipFree(dev_bt);
  free(t0);
  free(bt);
}
